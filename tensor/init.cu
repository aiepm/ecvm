#include "hip/hip_runtime.h"
#include <ecvm/tensor/init.cuh>

__global__ auto initCurandStates(hiprandState *states, unsigned long seed, int rows, int cols) -> void {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int totalElements = rows * cols;
  if (idx < totalElements) {
    hiprand_init(seed, idx, 0, &states[idx]);
  }
}

__global__ auto generateRandomMatrix(float *matrix, hiprandState *states, int rows, int cols) -> void {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int totalElements = rows * cols;
  if (idx < totalElements) {
    matrix[idx] = hiprand_uniform(&states[idx]);
  }
}

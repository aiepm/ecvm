#include <cstdio>
#include <ecvm/device/init.cuh>

auto init_device() -> int {
  int deviceCount;
  auto err = hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
    std::printf("No cuda-compatible devices found.\n");
    return 1;
  }

  std::printf("Found %d device(s)\n", deviceCount);

  // Optionally print device properties
  for (int i = 0; i < deviceCount; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    std::printf("Device %d: %s\n", i, prop.name);
  }
  err = hipSetDevice(0);
  if (err != hipSuccess) {
    std::printf("Failed to set device: %s\n", hipGetErrorString(err));
    return 1;
  }
  return 0;
}

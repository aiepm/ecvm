#include <cstdio>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblaslt.h>
#include <hip/hip_fp16.h>
#include <ecvm/gemm/naive.cuh>
#include <ecvm/gemm/tiled.cuh>
#include <ecvm/device/init.cuh>
#include <ecvm/tensor/init.cuh>
#include <ecvm/tensor/ops.cuh>
#include <nvtx3/nvToolsExt.h>
#include <string>
using i64 = int64_t;

constexpr float EPS = 1e-2;

auto get_time() -> i64 {
  struct timespec ts;
  clock_gettime(CLOCK_MONOTONIC, &ts);
  return ts.tv_sec * 1000000 + ts.tv_nsec / 1000;
}

auto get_microseconds(hipEvent_t &start, hipEvent_t &stop) -> i64 {
  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);
  return ms * 1000;
}

auto matmul_naive_vs_cublas() -> int {
  nvtxRangePush("matmul");
  int n = 4096;
  int k = 4096;
  int m = 4096;

  int BLOCK_SIZE = 1024;
  int NUM_BLOCKS = (n * m + BLOCK_SIZE - 1) / BLOCK_SIZE;

  float *d_a, *d_b, *d_c_ref, *d_c;
  nvtxRangePush("matrix memory allocation");
  hipMalloc(&d_a, n * k * sizeof(float));
  hipMalloc(&d_b, k * m * sizeof(float));
  hipMalloc(&d_c_ref, n * m * sizeof(float));
  hipMalloc(&d_c, n * m * sizeof(float));
  nvtxRangePop();

  auto run = [&](int iter_num, std::string name, float eps) -> int {
    float alpha = 1.0, beta = 0.0;
    i64 total_matmul_time = 0, total_cublas_time = 0, total_coalesced_time = 0;

    hiprandState *s_a, *s_b;
    hipMalloc(&s_a, n * k * sizeof(hiprandState));
    hipMalloc(&s_b, k * m * sizeof(hiprandState));

    
    bool *d_res, h_res;
    hipMalloc(&d_res, sizeof(bool));

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    for (int iter=0; iter<iter_num; iter++) {
      initCurandStates<<<NUM_BLOCKS, BLOCK_SIZE>>>(s_a, time(nullptr), n, k);
      initCurandStates<<<NUM_BLOCKS, BLOCK_SIZE>>>(s_b, time(nullptr), k, m);

      generateRandomMatrix<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_a, s_a, n, k);
      generateRandomMatrix<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_b, s_b, k, m);

      hipDeviceSynchronize();

      hipEvent_t coalesced_start, coalesced_end;
      hipEventCreate(&coalesced_start);
      hipEventCreate(&coalesced_end);
      // gridDim stays the same
      dim3 gridDim(4096 / 32, 4096 / 32);
      // make blockDim 1-dimensional, but don't change number of threads
      dim3 blockDim(32 * 32);
      nvtxRangePush("gemm_coalesced execution");
      hipEventRecord(coalesced_start);
      gemm_tiled<32><<<gridDim, blockDim>>>(d_c, d_a, d_b, n, k, m);
      hipEventRecord(coalesced_end);
      nvtxRangePop();

      hipDeviceSynchronize();

      hipEvent_t naive_start, naive_end;
      hipEventCreate(&naive_start);
      hipEventCreate(&naive_end);

      nvtxRangePush("gemm_naive execution");
      hipEventRecord(naive_start);
      gemm_naive<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_c, d_a, d_b, n, k, m);
      hipEventRecord(naive_end);
      nvtxRangePop();

      hipDeviceSynchronize();
      
      hipEvent_t cublas_start, cublas_end;
      hipEventCreate(&cublas_start);
      hipEventCreate(&cublas_end);

      nvtxRangePush("cublas sgemm execution");
      hipEventRecord(cublas_start);
      hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_b, m, d_a, k, &beta, d_c_ref, m);
      hipEventRecord(cublas_end);
      nvtxRangePop();

      hipDeviceSynchronize();

      total_matmul_time += get_microseconds(naive_start, naive_end);
      total_cublas_time += get_microseconds(cublas_start, cublas_end);
      total_coalesced_time += get_microseconds(coalesced_start, coalesced_end);

      h_res = true;
      hipMemcpy(d_res, &h_res, sizeof(bool), hipMemcpyHostToDevice);
      check_matrix_equality_atomic<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_c, d_c_ref, n, m, d_res, eps);
      hipDeviceSynchronize();
      hipMemcpy(&h_res, d_res, sizeof(bool), hipMemcpyDeviceToHost);

      if (!h_res) {
        return 1;
      }
    }

    i64 average_matmul_time = total_matmul_time / iter_num;
    i64 average_cublas_time = total_cublas_time / iter_num;
    i64 average_coalesced_time = total_coalesced_time / iter_num;
    double average_matmul_flops = ((2.0 * n * m * k) / average_matmul_time) / 1e3;
    double average_coalesced_flops = ((2.0 * n * m * k) / average_coalesced_time) / 1e3;

    std::printf("%s avg naive time: %ld\n", name.c_str(), average_matmul_time);
    std::printf("%s avg naive gflops: %lf\n", name.c_str(), average_matmul_flops);
    std::printf("%s avg coalesced time: %ld\n", name.c_str(), average_coalesced_time);
    std::printf("%s avg coalesced gflops: %lf\n", name.c_str(), average_coalesced_flops);
    std::printf("%s avg cublas time: %ld\n", name.c_str(), average_cublas_time);

    hipFree(s_a);
    hipFree(s_b);
    hipblasDestroy(handle);
    return 0;
  };

  int warmup_runs = 10;
  int perf_runs = 100;

  if(auto err = run(warmup_runs, "warmup", EPS); err != 0) {
    std::printf("Warmup failed\n");
    return err;
  }
  std::printf("Warmup successfull, naive = cublas\n");

  if (auto err = run(perf_runs, "perf", EPS); err != 0) {
    std::printf("Perf failed\n");
    return err;
  } 
  std::printf("Perf successfull, naive = cublas\n");

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  hipFree(d_c_ref);
  return 0;
}

auto main() -> int {
  if (auto err = init_device(); err != 0) {
    return err;
  }
  if (auto err = matmul_naive_vs_cublas(); err != 0) {
    return err;
  }
  return 0;
}

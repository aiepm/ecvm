#include <cstdio>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblaslt.h>
#include <hip/hip_fp16.h>
#include <ecvm/gemm/matmul_naive.cuh>
#include <ecvm/device/init.cuh>
#include <ecvm/tensor/init.cuh>
#include <ecvm/tensor/ops.cuh>
#include <nvtx3/nvToolsExt.h>
#include <string>
using i64 = int64_t;

constexpr float EPS = 1e-2;

auto get_time() -> i64 {
  struct timespec ts;
  clock_gettime(CLOCK_MONOTONIC, &ts);
  return ts.tv_sec * 1000000 + ts.tv_nsec / 1000;
}

auto get_microseconds(hipEvent_t &start, hipEvent_t &stop) -> i64 {
  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);
  return ms * 1000;
}

auto matmul_cpu_vs_gpu() -> int {
  nvtxRangePush("matmul");
  int n = 4092;
  int k = 4092;
  int m = 4092;

  int BLOCK_SIZE = 1024;
  int NUM_BLOCKS = (n * m + BLOCK_SIZE - 1) / BLOCK_SIZE;

  float *d_a, *d_b, *d_c_ref, *d_c;
  nvtxRangePush("matrix memory allocation");
  hipMalloc(&d_a, n * k * sizeof(float));
  hipMalloc(&d_b, k * m * sizeof(float));
  hipMalloc(&d_c_ref, n * m * sizeof(float));
  hipMalloc(&d_c, n * m * sizeof(float));
  nvtxRangePop();

  auto fmatmul_naive = [&](i64 &timer, hipStream_t &stream) -> int {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    nvtxRangePush("matmul_naive execution");
    hipEventRecord(start, stream);
    matmul_gpu<<<NUM_BLOCKS, BLOCK_SIZE, 0, stream>>>(d_c, d_a, d_b, n, k, m);
    hipEventRecord(stop, stream);
    nvtxRangePop();
    hipEventSynchronize(stop);
    timer += get_microseconds(start, stop);
    return 0;
  };

  auto fmatmul_cublas = [&](i64 &timer, hipStream_t &stream) -> int {
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetStream(handle, stream);

    float alpha = 1.0, beta = 0.0;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    nvtxRangePush("cublas sgemm execution");
    hipEventRecord(start, stream);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_b, m, d_a, k, &beta, d_c_ref, m);
    hipEventRecord(stop, stream);
    nvtxRangePop();
    hipEventSynchronize(stop);

    timer += get_microseconds(start, stop);
    hipblasDestroy(handle);
    return 0;
  };

  auto run = [&](int iter_num, std::string name, float eps) -> int {
    i64 total_matmul_time = 0;
    i64 total_cublas_time = 0;

    hiprandState *s_a, *s_b;
    hipMalloc(&s_a, n * k * sizeof(hiprandState));
    hipMalloc(&s_b, k * m * sizeof(hiprandState));

    hipStream_t s1, s2;
    hipStreamCreate(&s1);
    hipStreamCreate(&s2);

    for (int iter=0; iter<iter_num; iter++) {
      initCurandStates<<<NUM_BLOCKS, BLOCK_SIZE>>>(s_a, time(nullptr), n, k);
      initCurandStates<<<NUM_BLOCKS, BLOCK_SIZE>>>(s_b, time(nullptr), k, m);

      generateRandomMatrix<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_a, s_a, n, k);
      generateRandomMatrix<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_b, s_b, k, m);

      fmatmul_naive(total_matmul_time, s1);
      fmatmul_cublas(total_cublas_time, s2);

      hipStreamSynchronize(s1);
      hipStreamSynchronize(s2);

      bool *d_res, h_res;
      h_res = true;
      hipMalloc(&d_res, sizeof(bool));
      hipMemcpy(d_res, &h_res, sizeof(bool), hipMemcpyHostToDevice);
      check_matrix_equality_atomic<<<NUM_BLOCKS, BLOCK_SIZE, 0, s1>>>(d_c, d_c_ref, n, m, d_res, eps);
      hipStreamSynchronize(s1);
      hipMemcpy(&h_res, d_res, sizeof(bool), hipMemcpyDeviceToHost);

      if (!h_res) {
        return 1;
      }
    }
    
    i64 average_matmul_time = total_matmul_time / iter_num;
    i64 average_cublas_time = total_cublas_time / iter_num;

    std::printf("%s avg matmul time: %ld\n", name.c_str(), average_matmul_time);
    std::printf("%s avg cublas time: %ld\n", name.c_str(), average_cublas_time);

    hipFree(s_a);
    hipFree(s_b);
    hipStreamDestroy(s1);
    hipStreamDestroy(s2);
    return 0;
  };

  int warmup_runs = 10;
  int perf_runs = 100;

  if(auto err = run(warmup_runs, "warmup", EPS); err != 0) {
    std::printf("Warmup failed\n");
    return err;
  }
  std::printf("Warmup successfull, naive = cublas\n");

  if (auto err = run(perf_runs, "perf", EPS); err != 0) {
    std::printf("Perf failed\n");
    return err;
  } 
  std::printf("Perf successfull, naive = cublas\n");

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  hipFree(d_c_ref);
  return 0;
}

auto main() -> int {
  if (auto err = init_device(); err != 0) {
    return err;
  }
  if (auto err = matmul_cpu_vs_gpu(); err != 0) {
    return err;
  }
  return 0;
}
